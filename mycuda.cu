#include <stdio.h>
#include <hip/hip_runtime.h>

//#include <cutil_inline.h>

extern "C"
void runCudaPart(float a[], float b[], float c[], int n);


__global__ void myKernel(float *a, float *b, float *c, int n)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    //return;

    if (idx < n)
    {
        c[idx] = a[idx] + b[idx];
        //c[idx] = 10.0;
    }
    //printf("Test\n");
    return;
}

// Main cuda function

void runCudaPart(float *a, float *b, float *c, int n) {

    // all your cuda code here *smile*
    
    float *a_d, *b_d, *c_d;
    size_t size = n * sizeof(float);
    
    printf("n = %d\n", n);
    printf("%f %f %f\n", a[10], b[10], c[10]);

    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);

    //cudaMemcpy(a_d, a, size,cudaMemcpyHostToDevice);
    //cudaMemcpy(b_d, b, size,cudaMemcpyHostToDevice);
    //cudaMemset(c_d, 0, n);

    printf("Executing CUDA kernel\n");
    //myKernel <<<1,100>>> (a_d, b_d, c_d, n);



    hipMemcpy(c, c_d, size, hipMemcpyDeviceToHost);

    printf("Kernel ended.\n");

}
