#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "kernel.cuh"


#define N 100
#define MAX_ITER 500

__device__ float min_d(float a, float b)
{
	return (a<b? a : b);
}

__global__ void kernel(int *a, int *b, int *c)
{
	int globalID = threadIdx.x + blockIdx.x * blockDim.x;
	printf("globalID = %d\n", globalID);
	if (globalID < N)
		c[globalID] = a[globalID] + b[globalID];

}

//__global__ void Mandelbrot_kernel(uchar4* ptr, int width, int height, float* palette_d)
__global__ void Mandelbrot_kernel(uchar4* ptr, int width, int height, float* palette_d, float xmin, float xmax, float ymin, float ymax)
{
    // map from threadIdx/BlockIdx to pixel position
    float x = threadIdx.x + blockIdx.x * blockDim.x;
    float y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    
	  //float cx = 3.0f * (x / width-0.5f);
	  //float cy = 2.0f * (y / height-0.5f);


	  float cx = xmin + (x / width) * (xmax - xmin);
	  float cy = ymin + (y / height) * (ymax - ymin);


	//  int i;
 //   float zx = cx;
 //   float zy = cy;
	//	float dx = 0.0f;
	//	float dy = 0.0f; 
 //   //cx *=  cos(time/1000.0f);
 //   //cy *=  sin(time/1000.0f);
	////cx *= 1.0;
	////cy = 0.0;

	//  for(i=0; i<MAX_ITER; i++) {
	//		dx = (zx * zx - zy * zy) + cx + dx;
	//		dy = (zy * zx + zx * zy) + cy + dy;
	//		if((dx * dx + dy * dy) > 4.0f) break;
	//		zx = dx;
	//		zy = dy;
	//  }

			x = 0;
			y = 0;
			int iterNum = 0;
			float xtemp;

			while ( x*x + y*y < 4  &  iterNum < MAX_ITER)
			{
				xtemp = x*x - y*y + cx;
				y = 2*x*y + cy;
				x = xtemp;
				iterNum ++;
			}


	  //unsigned char val = 255.0f*float(i)/MAX_ITER;
	int val = 255.0f*(float)iterNum/MAX_ITER;
	  //printf("offset = %d, i = %d val = %d\n", offset, i, val);

    //
    //ptr[offset].x = min_d(255,3*val/2);
    //ptr[offset].y = (val>300 ? min_d(255,3*(val-100)) : 0);
    //ptr[offset].z = (val>500 ? min_d(255,2*(val-127)) : 0);
//ptr[offset].x = val;
//ptr[offset].y = MAX_ITER - val;
//ptr[offset].z = val*val%255;

ptr[offset].x = palette_d[val]*255;
ptr[offset].y = palette_d[val + 255]*255;
ptr[offset].z = palette_d[val + 510]*255;

//ptr[offset].x = val;
//ptr[offset].y = 255 - val;
//ptr[offset].z = palette_d[val];

    ptr[offset].w = 255;

	//if (offset < 1000)
	//	printf("offset = %d, i = %d val = %d palette.x = %f ptr.x = %f\n", offset, i, val, palette_d[val], ptr[offset].x);

}

extern "C" void launch_kernel()
{
	printf("RUN CUDA KERNEL\n");


	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void **) &dev_a, N*sizeof(int));
	hipMalloc((void **) &dev_b, N*sizeof(int));
	hipMalloc((void **) &dev_c, N*sizeof(int));
	// Fill Arrays
	for (int i = 0; i < N; i++)
	{
		a[i] = i,
			b[i] = 1;
	}
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	kernel<<<N,1>>>(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

}

//extern "C" void launch_Mandelbrot_kernel(uchar4* ptr, int width, int height)
extern "C" void launch_Mandelbrot_kernel(uchar4* ptr, int width, int height, float xmin, float xmax, float ymin, float ymax)
{
	dim3    blocks(width/16, height/16);
	dim3    threads(16,16);

	// set palette according to http://wojanton.ovh.org/ang/fractals/mandelbrotCpp.php

	float* palette;
	palette = (float*)malloc(255*3*sizeof(float));

for(int i=0; i < 64; i++){
	palette[i + 0] = (4*i)/(float)255;
	palette[i + 255] = (128-2*i)/(float)255;
	palette[i + 510] = (255-4*i)/(float)255;
}
for(int i=0; i < 64; i++){
	palette[64+i + 0] = (float)1;
	palette[64+i + 255] = (4*i)/(float)255;
	palette[64+i + 510] = (float)0;
}
for(int i=0; i < 64; i++){
	palette[128+i + 0] = (128-2*i)/(float)255;
	palette[128+i + 255] = (float)1;
	palette[128+i + 510] = (4*i)/(float)255;
}
for(int i=0; i < 64; i++){
	palette[192+i + 0] = (float)0;
	palette[192+i + 255] = (255-4*i)/(float)255;
	palette[192+i + 510] = (4*i)/(float)255;
}

	float* palette_d;
	hipMalloc((void **) &palette_d, 255*3*sizeof(float));	
	hipMemcpy(palette_d, palette, 255*3*sizeof(float), hipMemcpyHostToDevice);

	//Mandelbrot_kernel<<<blocks, threads>>>(ptr, width, height, palette_d);
	Mandelbrot_kernel<<<blocks, threads>>>(ptr, width, height, palette_d, xmin, xmax, ymin, ymax);
}